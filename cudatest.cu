#include "stdio.h"
#include "stdlib.h"
#include "hip/hip_runtime.h"

#include "hipblas.h"

#define BLOCK_HEIGHT 64 

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void switchRows(float* A, float* d_rowk, float* d_rowi, int* rightColumnIndices, int dim, int i, int k){
	int j = blockIdx.x*blockDim.x+threadIdx.x;
	int j_unshifted = blockIdx.x*blockDim.x+threadIdx.x;
	int temp_indexi;
	int temp_indexk;
	if (k!=i){
		temp_indexi=rightColumnIndices[i];
		temp_indexk=rightColumnIndices[k];
		rightColumnIndices[i]=temp_indexk;
		rightColumnIndices[k]=temp_indexi;
	}
	if (j+i>dim-1){
		j=dim-i+rightColumnIndices[j-dim+i];
	}
	A[(j+i)*dim+k]=d_rowk[j_unshifted]; // I store the line I am not interested in now
	A[(j+i)*dim+i]=d_rowi[j_unshifted];
	if (k!=i){
		A[(dim+temp_indexi)*dim+i]=0;
		A[(dim+temp_indexi)*dim+k]=1;
		A[(dim+temp_indexk)*dim+i]=1;
		A[(dim+temp_indexk)*dim+k]=0;
	}
}

__global__ void createIndexVector(int* rightColumnIndices, int dim){
	int j = blockIdx.x*blockDim.x+threadIdx.x;
	rightColumnIndices[j]=j;
}

__global__ void storeRows(float* A, float* d_rowk, float* d_rowi, int* rightColumnIndices, int dim, int i, int k){
	int j = blockIdx.x*blockDim.x+threadIdx.x;
	int j_unshifted = blockIdx.x*blockDim.x+threadIdx.x;
	if (j+i>dim-1){
		j=dim-i+rightColumnIndices[j-dim+i];
	}
	d_rowk[j_unshifted]=A[(j+i)*dim+i];
	d_rowi[j_unshifted]=A[(j+i)*dim+k];
}

__global__ void workRow(float* A, int* rightColumnIndices, int dim, int i){
	__shared__ float Aii;
	__shared__ float rowi[BLOCK_HEIGHT]; //tohle muzes vyhnat az na 1024!!!
	Aii=A[i*dim+i];
	int j = blockIdx.x*blockDim.x+threadIdx.x;
	int j0 =threadIdx.x;
	if (j+i>dim-2){
		j=dim-i-1+rightColumnIndices[j-dim+i+1];
	}
	rowi[j0]=A[(j+i+1)*dim+i];
	__syncthreads();
	rowi[j0]=rowi[j0]/Aii;
	A[(j+i+1)*dim+i]=rowi[j0];
/*	if (abs(Aii)<0.001) printf("%f\n", Aii);*/
}


__global__ void workRows(float* A, int* rightColumnIndices, int dim, int ipiv){
	__shared__ float colpiv[BLOCK_HEIGHT];
	__shared__ float colj[BLOCK_HEIGHT];
	__shared__ float colj_piv;
	int i = blockIdx.y*blockDim.x+threadIdx.x;
	int i0 = threadIdx.x;
	int j = blockIdx.x;
	if (j+ipiv>dim-2){
		j=dim-ipiv-1+rightColumnIndices[j-dim+ipiv+1];
	}
	colpiv[i0]=A[ipiv*dim+i];
	colj[i0]=A[(j+ipiv+1)*dim+i];
	colj_piv=A[(j+ipiv+1)*dim+ipiv];
	if (i != ipiv){
		colj[i0]=colj[i0]-colj_piv*colpiv[i0];
	}
	A[(j+ipiv+1)*dim+i]=colj[i0];
}

__global__ void createPivotVector(float* A, float* v, int dim, int ipiv){
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<ipiv){
		v[i]=0;
	} else {
		v[i]=A[ipiv*dim+i];
	}
}

extern "C" void kernel_wrapper_(float* A, int* dim){
	size_t size=2**dim**dim*sizeof(float);
	size_t sizecol=*dim*sizeof(float);
	size_t sizerow=*dim*sizeof(float);
	size_t sizerowint=*dim*sizeof(int);
	float* d_A;
	float* d_col;
	float* d_rowi;
	float* d_rowk;
	int* d_indices;
	int max_idx;
	int strong_pivoting=1;
	hipblasHandle_t handle;
	hipblasStatus_t stat;

	hipMalloc(&d_A,size);
	hipMalloc(&d_col,sizecol);
	hipMalloc(&d_rowi,sizerow);
	hipMalloc(&d_rowk,sizerow);
	hipMalloc(&d_indices,sizerowint);
	gpuErrchk( hipMemcpy(d_A,A, size,hipMemcpyHostToDevice) );
	hipblasCreate(&handle);
	
	dim3 dimBlockRow(BLOCK_HEIGHT,1,1);
	dim3 dimGridRow(*dim/BLOCK_HEIGHT,1,1);
	dim3 dimBlockCol(BLOCK_HEIGHT,1,1);
	dim3 dimGridCol(*dim,*dim/BLOCK_HEIGHT,1);
	dim3 dimBlockPiv(BLOCK_HEIGHT,1,1);
	dim3 dimGridPiv(*dim/BLOCK_HEIGHT,1,1);
	dim3 dimBlockStoreRow(BLOCK_HEIGHT,1,1);
	dim3 dimGridStoreRow(*dim/BLOCK_HEIGHT,1,1);
	createIndexVector<<<dimGridRow,dimBlockRow>>>(d_indices,*dim);
	for (int i=0; i<*dim;++i){
		if (strong_pivoting == 1){
			createPivotVector<<<dimGridPiv,dimBlockPiv>>>(d_A,d_col,*dim,i);
			stat = hipblasIsamax(handle, *dim, d_col, 1, &max_idx);
			if (stat != HIPBLAS_STATUS_SUCCESS) printf("Max failed\n");
			storeRows<<<dimGridStoreRow,dimBlockStoreRow>>>(d_A,d_rowk,d_rowi,d_indices,*dim,i,max_idx-1);
			switchRows<<<dimGridStoreRow,dimBlockStoreRow>>>(d_A,d_rowk,d_rowi,d_indices,*dim,i,max_idx-1);
		} else{
        	max_idx=i+1;
		}
		workRow<<<dimGridRow,dimBlockRow>>>(d_A,d_indices,*dim,i);
		workRows<<<dimGridCol,dimBlockCol>>>(d_A,d_indices,*dim,i);
	}
    hipblasDestroy(handle);
	gpuErrchk( hipMemcpy(A,d_A, size,hipMemcpyDeviceToHost) );
	gpuErrchk( hipFree(d_A) );
	gpuErrchk( hipFree(d_col) );
	gpuErrchk( hipFree(d_rowi) );
	gpuErrchk( hipFree(d_rowk) );
	gpuErrchk( hipFree(d_indices) );

	return;

}