#include "stdio.h"
#include "stdlib.h"
#include "hip/hip_runtime.h"

#include "hipblas.h"

/* a clumsy way to define block size, is not a variable */
#define BLOCK_HEIGHT 64 

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


__global__ void switchRows(float* A, float* d_rowk, float* d_rowi, int* rightColumnIndices, int dim, int i, int k){
	/* this subroutine switches two lines according to the partial pivoting */
	int j = blockIdx.x*blockDim.x+threadIdx.x;
	int j_unshifted = blockIdx.x*blockDim.x+threadIdx.x;
	int temp_indexi;
	int temp_indexk;
	if (k!=i){
		temp_indexi=rightColumnIndices[i];
		temp_indexk=rightColumnIndices[k];
		rightColumnIndices[i]=temp_indexk;
		rightColumnIndices[k]=temp_indexi;
	}
	if (j+i>dim-1){
		j=dim-i+rightColumnIndices[j-dim+i];
	}
	A[(j+i)*dim+k]=d_rowk[j_unshifted]; // store the line I am not interested in now
	A[(j+i)*dim+i]=d_rowi[j_unshifted];
	if (k!=i){
		A[(dim+temp_indexi)*dim+i]=0;
		A[(dim+temp_indexi)*dim+k]=1;
		A[(dim+temp_indexk)*dim+i]=1;
		A[(dim+temp_indexk)*dim+k]=0;
	}
}

__global__ void createIndexVector(int* rightColumnIndices, int dim){
	/* this subroutine creates a vector with indices to keep track of where */
	/* ones are in the RHS matrix */
	int j = blockIdx.x*blockDim.x+threadIdx.x;
	rightColumnIndices[j]=j;
}

__global__ void storeRows(float* A, float* d_rowk, float* d_rowi, int* rightColumnIndices, int dim, int i, int k){
	/* this subroutine  stores two rows into auxiliary variables in order to switch them with switchRows() */
	int j = blockIdx.x*blockDim.x+threadIdx.x;
	int j_unshifted = blockIdx.x*blockDim.x+threadIdx.x;
	if (j+i>dim-1){
		j=dim-i+rightColumnIndices[j-dim+i];
	}
	d_rowk[j_unshifted]=A[(j+i)*dim+i];
	d_rowi[j_unshifted]=A[(j+i)*dim+k];
}

__global__ void workRow(float* A, int* rightColumnIndices, int dim, int i){
	/* divides the whole row by pivot's value */
	__shared__ float Aii;
	__shared__ float rowi[BLOCK_HEIGHT]; 
	Aii=A[i*dim+i];
	int j = blockIdx.x*blockDim.x+threadIdx.x;
	int j0 =threadIdx.x;
	if (j+i>dim-2){
		j=dim-i-1+rightColumnIndices[j-dim+i+1];
	}
	rowi[j0]=A[(j+i+1)*dim+i];
	__syncthreads();
	rowi[j0]=rowi[j0]/Aii;
	A[(j+i+1)*dim+i]=rowi[j0];
}


__global__ void workRows(float* A, int* rightColumnIndices, int dim, int ipiv){
	/* subtracts the pivot row from other rows */
	__shared__ float colpiv[BLOCK_HEIGHT];
	__shared__ float colj[BLOCK_HEIGHT];
	__shared__ float colj_piv;
	int i = blockIdx.y*blockDim.x+threadIdx.x;
	int i0 = threadIdx.x;
	int j = blockIdx.x;
	if (j+ipiv>dim-2){
		j=dim-ipiv-1+rightColumnIndices[j-dim+ipiv+1];
	}
	colpiv[i0]=A[ipiv*dim+i];
	colj[i0]=A[(j+ipiv+1)*dim+i];
	colj_piv=A[(j+ipiv+1)*dim+ipiv];
	if (i != ipiv){
		colj[i0]=colj[i0]-colj_piv*colpiv[i0];
	}
	A[(j+ipiv+1)*dim+i]=colj[i0];
}

__global__ void createPivotVector(float* A, float* v, int dim, int ipiv){
	/* this subroutine saves a column used for pivoting into an aux variable */
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if (i<ipiv){
		v[i]=0;
	} else {
		v[i]=A[ipiv*dim+i];
	}
}

extern "C" void kernel_wrapper_(float* A, int* dim){
	size_t size=2**dim**dim*sizeof(float);
	size_t sizecol=*dim*sizeof(float);
	size_t sizerow=*dim*sizeof(float);
	size_t sizerowint=*dim*sizeof(int);
	float* d_A;
	float* d_col;
	float* d_rowi;
	float* d_rowk;
	int* d_indices;
	int max_idx;
	int partial_pivoting=1;
	/* cublas is blas for cuda, used here only to find the pivot */
	hipblasHandle_t handle;
	hipblasStatus_t stat;
	
	/* allocate the varaibles that I'll need on the GPU */
	hipMalloc(&d_A,size);
	hipMalloc(&d_col,sizecol);
	hipMalloc(&d_rowi,sizerow);
	hipMalloc(&d_rowk,sizerow);
	hipMalloc(&d_indices,sizerowint);
	/* copy the matrix to gpu */
	gpuErrchk( hipMemcpy(d_A,A, size,hipMemcpyHostToDevice) );
	hipblasCreate(&handle);

	/* define dimensions */	
	dim3 dimBlockRow(BLOCK_HEIGHT,1,1);
	dim3 dimGridRow(*dim/BLOCK_HEIGHT,1,1);
	dim3 dimBlockCol(BLOCK_HEIGHT,1,1);
	dim3 dimGridCol(*dim,*dim/BLOCK_HEIGHT,1);
	dim3 dimBlockPiv(BLOCK_HEIGHT,1,1);
	dim3 dimGridPiv(*dim/BLOCK_HEIGHT,1,1);
	dim3 dimBlockStoreRow(BLOCK_HEIGHT,1,1);
	dim3 dimGridStoreRow(*dim/BLOCK_HEIGHT,1,1);
	createIndexVector<<<dimGridRow,dimBlockRow>>>(d_indices,*dim);
	for (int i=0; i<*dim;++i){
		if (partial_pivoting == 1){
			/* it is not really partially pivoting... I search for the pivot only in the rows */
			/* that I didn't process yet -- this is only because of laziness. It can be improved, */ 
			/* but the row switching would be more complicated. */
			createPivotVector<<<dimGridPiv,dimBlockPiv>>>(d_A,d_col,*dim,i);
			stat = hipblasIsamax(handle, *dim, d_col, 1, &max_idx);
			if (stat != HIPBLAS_STATUS_SUCCESS) printf("Max failed\n");
			storeRows<<<dimGridStoreRow,dimBlockStoreRow>>>(d_A,d_rowk,d_rowi,d_indices,*dim,i,max_idx-1);
			switchRows<<<dimGridStoreRow,dimBlockStoreRow>>>(d_A,d_rowk,d_rowi,d_indices,*dim,i,max_idx-1);
		} else{
        	max_idx=i+1;
		}
		workRow<<<dimGridRow,dimBlockRow>>>(d_A,d_indices,*dim,i);
		workRows<<<dimGridCol,dimBlockCol>>>(d_A,d_indices,*dim,i);
	}
   	hipblasDestroy(handle);
	/* get the result from the GPU */
	gpuErrchk( hipMemcpy(A,d_A, size,hipMemcpyDeviceToHost) );
	/* cleanup */
	gpuErrchk( hipFree(d_A) );
	gpuErrchk( hipFree(d_col) );
	gpuErrchk( hipFree(d_rowi) );
	gpuErrchk( hipFree(d_rowk) );
	gpuErrchk( hipFree(d_indices) );

	return;

}
